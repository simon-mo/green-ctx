#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define IPC_HANDLE_TYPE hipMemHandleTypePosixFileDescriptor

void checkCudaErrors(hipError_t err, const char* msg) {
    if(err != hipSuccess) {
        const char *errStr = nullptr;
        hipDrvGetErrorString(err, &errStr);
        fprintf(stderr, "Error: %s: %s\n", msg, errStr ? errStr : "Unknown error");
        exit(EXIT_FAILURE);
    }
}

int main() {
    checkCudaErrors(hipInit(0), "hipInit");
    hipDevice_t cuDevice;
    checkCudaErrors(hipDeviceGet(&cuDevice, 0), "hipDeviceGet");
    hipCtx_t cuContext;
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice), "hipCtxCreate");

    size_t allocSize = 10 * 1024 * 1024; // 10 MB

    hipMemAllocationProp allocProp = {};
    allocProp.type = hipMemAllocationTypePinned;
    allocProp.location.type = hipMemLocationTypeDevice;
    allocProp.location.id = 0;
    allocProp.requestedHandleTypes = IPC_HANDLE_TYPE;

    size_t granularity = 0;
    checkCudaErrors(hipMemGetAllocationGranularity(&granularity, &allocProp, hipMemAllocationGranularityMinimum),
                    "hipMemGetAllocationGranularity");
    if (allocSize % granularity != 0) {
        allocSize = ((allocSize + granularity - 1) / granularity) * granularity;
    }

    hipMemGenericAllocationHandle_t memHandle;
    checkCudaErrors(hipMemCreate(&memHandle, allocSize, &allocProp, 0), "hipMemCreate");
    printf("Allocated physical memory of size %zu bytes\n", allocSize);

    hipDeviceptr_t basePtr = 0;
    checkCudaErrors(hipMemAddressReserve(&basePtr, allocSize, 0, 0, 0), "hipMemAddressReserve");
    checkCudaErrors(hipMemMap(basePtr, allocSize, 0, memHandle, 0), "hipMemMap");

    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    checkCudaErrors(hipMemSetAccess(basePtr, allocSize, &accessDesc, 1), "hipMemSetAccess");
    printf("Server: Mapped physical memory to virtual address %p\n", (void*)basePtr);

    double initData[2] = {3.1415, 2.71828};
    checkCudaErrors(hipMemcpyHtoD(basePtr, initData, 2 * sizeof(double)), "hipMemcpyHtoD");
    printf("Server: Wrote initial data: %.4f, %.4f\n", initData[0], initData[1]);

    int shareableHandle;
    checkCudaErrors(hipMemExportToShareableHandle(&shareableHandle, memHandle, IPC_HANDLE_TYPE, 0),
                    "hipMemExportToShareableHandle");
    printf("Server: Exported shareable handle: %d\n", shareableHandle);

    hipMemGenericAllocationHandle_t importedHandle;
    checkCudaErrors(hipMemImportFromShareableHandle(&importedHandle,
                  (void*)(uintptr_t)shareableHandle, IPC_HANDLE_TYPE),
                  "hipMemImportFromShareableHandle");
    printf("Client: Imported memory handle successfully\n");
    hipDeviceptr_t clientBasePtr = 0;
    checkCudaErrors(hipMemAddressReserve(&clientBasePtr, allocSize, 0, 0, 0), "hipMemAddressReserve (client)");
    checkCudaErrors(hipMemMap(clientBasePtr, allocSize, 0, importedHandle, 0), "hipMemMap (client)");

    hipMemAccessDesc clientAccessDesc = {};
    clientAccessDesc.location.type = hipMemLocationTypeDevice;
    clientAccessDesc.location.id = 0;
    clientAccessDesc.flags = hipMemAccessFlagsProtReadWrite;
    checkCudaErrors(hipMemSetAccess(clientBasePtr, allocSize, &clientAccessDesc, 1), "hipMemSetAccess (client)");
    printf("Client: Mapped imported memory to virtual address %p\n", (void*)clientBasePtr);

    double clientData[2] = {0.0, 0.0};
    checkCudaErrors(hipMemcpyDtoH(clientData, clientBasePtr, 2 * sizeof(double)), "hipMemcpyDtoH (client)");
    printf("Client: Read data: %.4f, %.4f\n", clientData[0], clientData[1]);

    checkCudaErrors(hipMemUnmap(basePtr, allocSize), "hipMemUnmap (server)");
    checkCudaErrors(hipMemAddressFree(basePtr, allocSize), "hipMemAddressFree (server)");

    checkCudaErrors(hipMemUnmap(clientBasePtr, allocSize), "hipMemUnmap (client)");
    checkCudaErrors(hipMemAddressFree(clientBasePtr, allocSize), "hipMemAddressFree (client)");

    checkCudaErrors(hipMemRelease(memHandle), "hipMemRelease");

    checkCudaErrors(hipCtxDestroy(cuContext), "hipCtxDestroy");

    printf("Program completed successfully.\n");
    return 0;
}
