#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <hip/hip_runtime.h>

#define SOCKET_PATH "/tmp/cuda_ipc_socket"
#define IPC_HANDLE_TYPE hipMemHandleTypePosixFileDescriptor

void checkCudaErrors(hipError_t err, const char* msg) {
    if(err != hipSuccess) {
        const char* errStr = nullptr;
        hipDrvGetErrorString(err, &errStr);
        fprintf(stderr, "Server Error: %s: %s\n", msg, errStr ? errStr : "Unknown error");
        exit(EXIT_FAILURE);
    }
}

int send_fd(int socket, int fd_to_send) {
    struct msghdr msg = {0};
    char buf[CMSG_SPACE(sizeof(fd_to_send))];
    memset(buf, 0, sizeof(buf));
    
    struct iovec io = { .iov_base = (void*)"ABC", .iov_len = 3 };
    msg.msg_iov = &io;
    msg.msg_iovlen = 1;
    
    msg.msg_control = buf;
    msg.msg_controllen = sizeof(buf);
    struct cmsghdr* cmsg = CMSG_FIRSTHDR(&msg);
    cmsg->cmsg_level = SOL_SOCKET;
    cmsg->cmsg_type  = SCM_RIGHTS;
    cmsg->cmsg_len   = CMSG_LEN(sizeof(fd_to_send));
    memcpy(CMSG_DATA(cmsg), &fd_to_send, sizeof(fd_to_send));
    msg.msg_controllen = cmsg->cmsg_len;
    
    if (sendmsg(socket, &msg, 0) < 0) {
        perror("Server: sendmsg");
        return -1;
    }
    return 0;
}

int main() {
    checkCudaErrors(hipInit(0), "hipInit");
    hipDevice_t cuDevice;
    checkCudaErrors(hipDeviceGet(&cuDevice, 0), "hipDeviceGet");
    hipCtx_t cuContext;
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice), "hipCtxCreate");
    
    size_t allocSize = 15ULL * 1024ULL * 1024ULL * 1024ULL;
    hipMemAllocationProp allocProp = {};
    allocProp.type = hipMemAllocationTypePinned;

    allocProp.location.type = hipMemLocationTypeDevice;
    allocProp.location.id = 0;
    allocProp.requestedHandleTypes = IPC_HANDLE_TYPE;
    
    size_t granularity = 0;
    checkCudaErrors(hipMemGetAllocationGranularity(&granularity, &allocProp, hipMemAllocationGranularityMinimum),
                     "hipMemGetAllocationGranularity");
    if (allocSize % granularity != 0) {
        allocSize = ((allocSize + granularity - 1) / granularity) * granularity;
    }
    
    hipMemGenericAllocationHandle_t memHandle;
    checkCudaErrors(hipMemCreate(&memHandle, allocSize, &allocProp, 0), "hipMemCreate");
    printf("Server: Allocated physical memory of size %zu bytes\n", allocSize);

    hipDeviceptr_t basePtr = 0;
    checkCudaErrors(hipMemAddressReserve(&basePtr, allocSize, 0, 0, 0), "hipMemAddressReserve");
    checkCudaErrors(hipMemMap(basePtr, allocSize, 0, memHandle, 0), "hipMemMap");
    
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    checkCudaErrors(hipMemSetAccess(basePtr, allocSize, &accessDesc, 1), "hipMemSetAccess");
    printf("Server: Mapped memory to virtual address %p\n", (void*)basePtr);

    double initData[2] = {3.1415, 2.71828};
    checkCudaErrors(hipMemcpyHtoD(basePtr, initData, 2 * sizeof(double)), "hipMemcpyHtoD");
    printf("Server: Wrote initial data: %.4f, %.4f\n", initData[0], initData[1]);
    

    int shareableFd;
    checkCudaErrors(hipMemExportToShareableHandle(&shareableFd, memHandle, IPC_HANDLE_TYPE, 0),
                     "hipMemExportToShareableHandle"); 
    printf("Server: Exported shareable handle (fd): %d\n", shareableFd);
    
    int server_sock = socket(AF_UNIX, SOCK_STREAM, 0);
    if (server_sock < 0) {
        perror("Server: socket");
        exit(EXIT_FAILURE);
    }
    struct sockaddr_un addr;
    memset(&addr, 0, sizeof(addr));
    addr.sun_family = AF_UNIX;
    strncpy(addr.sun_path, SOCKET_PATH, sizeof(addr.sun_path) - 1);
    unlink(SOCKET_PATH);
    if (bind(server_sock, (struct sockaddr*)&addr, sizeof(addr)) < 0) {
        perror("Server: bind");
        exit(EXIT_FAILURE);
    }
    if (listen(server_sock, 1) < 0) {
        perror("Server: listen");
        exit(EXIT_FAILURE);
    }
    
    printf("Server: Waiting for client connection...\n");
    int client_sock = accept(server_sock, NULL, NULL);
    if (client_sock < 0) {
        perror("Server: accept");
        exit(EXIT_FAILURE);
    }
    printf("Server: Client connected, sending FD...\n");
    

    if (send_fd(client_sock, shareableFd) < 0) {
        fprintf(stderr, "Server: Failed to send FD\n");
        exit(EXIT_FAILURE);
    }
    printf("Server: FD sent successfully.\n");
    
    close(client_sock);
    close(server_sock);
    sleep(10);
    
    checkCudaErrors(hipMemUnmap(basePtr, allocSize), "hipMemUnmap");
    checkCudaErrors(hipMemAddressFree(basePtr, allocSize), "hipMemAddressFree");
    checkCudaErrors(hipMemRelease(memHandle), "hipMemRelease");
    checkCudaErrors(hipCtxDestroy(cuContext), "hipCtxDestroy");
    
    printf("Server: Cleanup complete, exiting.\n");
    return 0;
}
