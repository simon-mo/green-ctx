#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <hip/hip_runtime.h>

#define SOCKET_PATH "/tmp/cuda_ipc_socket"
#define IPC_HANDLE_TYPE hipMemHandleTypePosixFileDescriptor

void checkCudaErrors(hipError_t err, const char* msg) {
    if(err != hipSuccess) {
        const char* errStr = nullptr;
        hipDrvGetErrorString(err, &errStr);
        fprintf(stderr, "Client Error: %s: %s\n", msg, errStr ? errStr : "Unknown error");
        exit(EXIT_FAILURE);
    }
}

int recv_fd(int socket) {
    struct msghdr msg = {0};
    char m_buffer[256];
    struct iovec io = { .iov_base = m_buffer, .iov_len = sizeof(m_buffer) };
    msg.msg_iov = &io;
    msg.msg_iovlen = 1;
    
    char c_buffer[CMSG_SPACE(sizeof(int))];
    memset(c_buffer, 0, sizeof(c_buffer));
    msg.msg_control = c_buffer;
    msg.msg_controllen = sizeof(c_buffer);
    
    if (recvmsg(socket, &msg, 0) < 0) {
        perror("Client: recvmsg");
        return -1;
    }
    
    struct cmsghdr* cmsg = CMSG_FIRSTHDR(&msg);
    if (cmsg == NULL) {
        fprintf(stderr, "Client: No passed fd\n");
        return -1;
    }
    
    int fd;
    memcpy(&fd, CMSG_DATA(cmsg), sizeof(fd));
    return fd;
}

int main() {
    checkCudaErrors(hipInit(0), "hipInit");
    hipDevice_t cuDevice;
    checkCudaErrors(hipDeviceGet(&cuDevice, 0), "hipDeviceGet");
    hipCtx_t cuContext;
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice), "hipCtxCreate");
    
    int sock = socket(AF_UNIX, SOCK_STREAM, 0);
    if (sock < 0) {
        perror("Client: socket");
        exit(EXIT_FAILURE);
    }
    struct sockaddr_un addr;
    memset(&addr, 0, sizeof(addr));
    addr.sun_family = AF_UNIX;
    strncpy(addr.sun_path, SOCKET_PATH, sizeof(addr.sun_path)-1);
    if (connect(sock, (struct sockaddr*)&addr, sizeof(addr)) < 0) {
        perror("Client: connect");
        exit(EXIT_FAILURE);
    }
    printf("Client: Connected to server socket\n");
    
    int received_fd = recv_fd(sock);
    if (received_fd < 0) {
        fprintf(stderr, "Client: Failed to receive FD\n");
        exit(EXIT_FAILURE);
    }
    printf("Client: Received FD: %d\n", received_fd);
    
    close(sock);
    
    hipMemGenericAllocationHandle_t importedHandle;
    checkCudaErrors(hipMemImportFromShareableHandle(&importedHandle,
                  (void*)(uintptr_t)received_fd, IPC_HANDLE_TYPE),
                  "hipMemImportFromShareableHandle");
    printf("Client: Imported memory handle successfully\n");
    
    size_t allocSize = 10 * 1024 * 1024; 
    hipDeviceptr_t basePtr = 0;
    checkCudaErrors(hipMemAddressReserve(&basePtr, allocSize, 0, 0, 0), "hipMemAddressReserve");
    checkCudaErrors(hipMemMap(basePtr, allocSize, 0, importedHandle, 0), "hipMemMap");
    
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    checkCudaErrors(hipMemSetAccess(basePtr, allocSize, &accessDesc, 1), "hipMemSetAccess");
    printf("Client: Mapped imported memory to virtual address %p\n", (void*)basePtr);
    
    double data[2] = {0.0, 0.0};
    checkCudaErrors(hipMemcpyDtoH(data, basePtr, 2 * sizeof(double)), "hipMemcpyDtoH");
    printf("Client: Read initial data: %.4f, %.4f\n", data[0], data[1]);
    
    double newData[2] = { data[0] + 1.0, data[1] + 1.0 };
    checkCudaErrors(hipMemcpyHtoD(basePtr, newData, 2 * sizeof(double)), "hipMemcpyHtoD");
    printf("Client: Wrote new data: %.4f, %.4f\n", newData[0], newData[1]);
    
    double verifyData[2] = {0.0, 0.0};
    checkCudaErrors(hipMemcpyDtoH(verifyData, basePtr, 2 * sizeof(double)), "hipMemcpyDtoH");
    printf("Client: Verified data: %.4f, %.4f\n", verifyData[0], verifyData[1]);
    
    checkCudaErrors(hipMemUnmap(basePtr, allocSize), "hipMemUnmap");
    checkCudaErrors(hipMemAddressFree(basePtr, allocSize), "hipMemAddressFree");
    checkCudaErrors(hipCtxDestroy(cuContext), "hipCtxDestroy");
    
    printf("Client: Cleanup complete, exiting.\n");
    return 0;
}
