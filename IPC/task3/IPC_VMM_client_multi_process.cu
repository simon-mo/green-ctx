#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <sys/socket.h>
#include <sys/un.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#define SOCKET_PATH "/tmp/cuda_ipc_socket"
#define IPC_HANDLE_TYPE hipMemHandleTypePosixFileDescriptor

void checkCudaErrors(hipError_t err, const char* msg) {
    if(err != hipSuccess) {
        const char* errStr = nullptr;
        hipDrvGetErrorString(err, &errStr);
        fprintf(stderr, "Client Error: %s: %s\n", msg, errStr ? errStr : "Unknown error");
        exit(EXIT_FAILURE);
    }
}


int recv_fd(int socket) {
    struct msghdr msg = {0};
    char m_buffer[256];
    struct iovec io = { .iov_base = m_buffer, .iov_len = sizeof(m_buffer) };
    msg.msg_iov = &io;
    msg.msg_iovlen = 1;
    
    char c_buffer[CMSG_SPACE(sizeof(int))];
    memset(c_buffer, 0, sizeof(c_buffer));
    msg.msg_control = c_buffer;
    msg.msg_controllen = sizeof(c_buffer);
    
    if (recvmsg(socket, &msg, 0) < 0) {
        perror("Client: recvmsg");
        return -1;
    }
    
    struct cmsghdr* cmsg = CMSG_FIRSTHDR(&msg);
    if (cmsg == NULL) {
        fprintf(stderr, "Client: No passed FD\n");
        return -1;
    }
    
    int fd;
    memcpy(&fd, CMSG_DATA(cmsg), sizeof(fd));
    return fd;
}

void clientProcessWork(int procIndex, int sharedFd) {
    printf("Child process %d: Starting\n", procIndex);
    
    checkCudaErrors(hipInit(0), "hipInit");
    hipDevice_t cuDevice;
    checkCudaErrors(hipDeviceGet(&cuDevice, 0), "hipDeviceGet");
    hipCtx_t cuContext;
    checkCudaErrors(hipCtxCreate(&cuContext, 0, cuDevice), "hipCtxCreate");
    
    hipMemGenericAllocationHandle_t importedHandle;
    checkCudaErrors(hipMemImportFromShareableHandle(&importedHandle,
                  (void*)(uintptr_t)sharedFd, IPC_HANDLE_TYPE),
                  "hipMemImportFromShareableHandle");
    printf("Child process %d: Imported memory handle successfully\n", procIndex);
    
    // size_t allocSize = 10 * 1024 * 1024; // 10 MB
    size_t allocSize = 15ULL * 1024ULL * 1024ULL * 1024ULL;
    hipDeviceptr_t basePtr = 0;
    checkCudaErrors(hipMemAddressReserve(&basePtr, allocSize, 0, 0, 0),
                    "hipMemAddressReserve");
    checkCudaErrors(hipMemMap(basePtr, allocSize, 0, importedHandle, 0),
                    "hipMemMap");
    
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    checkCudaErrors(hipMemSetAccess(basePtr, allocSize, &accessDesc, 1),
                    "hipMemSetAccess");
    printf("Child process %d: Mapped imported memory to virtual address %p\n",
           procIndex, (void*)basePtr);
    
    double data[2] = {0.0, 0.0};
    checkCudaErrors(hipMemcpyDtoH(data, basePtr, 2 * sizeof(double)),
                    "hipMemcpyDtoH");
    printf("Child process %d: Read data: %.4f, %.4f\n", procIndex, data[0], data[1]);
    
    double newData[2] = { data[0] + procIndex, data[1] + procIndex };
    checkCudaErrors(hipMemcpyHtoD(basePtr, newData, 2 * sizeof(double)),
                    "hipMemcpyHtoD");
    printf("Child process %d: Wrote new data: %.4f, %.4f\n", procIndex, newData[0], newData[1]);
    
    double verifyData[2] = {0.0, 0.0};
    checkCudaErrors(hipMemcpyDtoH(verifyData, basePtr, 2 * sizeof(double)),
                    "hipMemcpyDtoH verify");
    printf("Child process %d: Verified data: %.4f, %.4f\n", procIndex, verifyData[0], verifyData[1]);
    
    checkCudaErrors(hipMemUnmap(basePtr, allocSize), "hipMemUnmap");
    checkCudaErrors(hipMemAddressFree(basePtr, allocSize), "hipMemAddressFree");
    checkCudaErrors(hipCtxDestroy(cuContext), "hipCtxDestroy");
    
    printf("Child process %d: Exiting\n", procIndex);
    exit(EXIT_SUCCESS);
}

int main() {
    int sock = socket(AF_UNIX, SOCK_STREAM, 0);
    if (sock < 0) {
        perror("Client (parent): socket");
        exit(EXIT_FAILURE);
    }
    struct sockaddr_un addr;
    memset(&addr, 0, sizeof(addr));
    addr.sun_family = AF_UNIX;
    strncpy(addr.sun_path, SOCKET_PATH, sizeof(addr.sun_path)-1);
    if (connect(sock, (struct sockaddr*)&addr, sizeof(addr)) < 0) {
        perror("Client (parent): connect");
        exit(EXIT_FAILURE);
    }
    printf("Client (parent): Connected to server socket\n");
    
    int sharedFd = recv_fd(sock);
    if (sharedFd < 0) {
        fprintf(stderr, "Client (parent): Failed to receive FD\n");
        exit(EXIT_FAILURE);
    }
    printf("Client (parent): Received shared FD: %d\n", sharedFd);
    close(sock);
    
    int numClients = 3;
    pid_t pids[numClients];
    for (int i = 0; i < numClients; i++) {
        pid_t pid = fork();
        if (pid < 0) {
            perror("Client (parent): fork");
            exit(EXIT_FAILURE);
        }
        if (pid == 0) {
            int dupFd = dup(sharedFd);
            if (dupFd < 0) {
                perror("Client (child): dup");
                exit(EXIT_FAILURE);
            }
            clientProcessWork(i, dupFd);
        } else {
            pids[i] = pid;
        }
    }
    
    for (int i = 0; i < numClients; i++) {
        int status;
        waitpid(pids[i], &status, 0);
        printf("Client (parent): Child %d exited with status %d\n", i, status);
    }
    
    close(sharedFd); 
    printf("Client (parent): All child processes completed.\n");
    return 0;
}
