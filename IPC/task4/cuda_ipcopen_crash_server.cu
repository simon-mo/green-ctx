#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>

// Function to check CUDA errors
void checkCuda(hipError_t err, const char* msg) {
    if(err != hipSuccess) {
        fprintf(stderr, "Server Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Helper function to write IPC handle to a file
void writeIpcHandleToFile(const char* filename, hipIpcMemHandle_t* handle) {
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        perror("Server: fopen ipc_handle.bin");
        exit(EXIT_FAILURE);
    }
    fwrite(handle, sizeof(hipIpcMemHandle_t), 1, fp);
    fclose(fp);
    printf("Server: IPC handle written to %s\n", filename);
}

int main() {
    const int numElements = 1000;
    // Allocate GPU memory for an array of 1000 floats
    float* d_ptr = nullptr;
    checkCuda(hipMalloc(&d_ptr, numElements * sizeof(float)), "hipMalloc");

    // Initialize the array with known values: h_data[i] = i * 1.0f
    float* h_data = new float[numElements];
    for (int i = 0; i < numElements; i++) {
        h_data[i] = static_cast<float>(i);
    }
    checkCuda(hipMemcpy(d_ptr, h_data, numElements * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy init");

    printf("Server: GPU memory allocated at %p, capacity = %d floats\n", d_ptr, numElements);

    // Obtain the IPC handle for the allocated memory
    hipIpcMemHandle_t ipcHandle;
    checkCuda(hipIpcGetMemHandle(&ipcHandle, d_ptr), "hipIpcGetMemHandle");

    // Write the IPC handle to a file so that the client can open it
    writeIpcHandleToFile("ipc_handle.bin", &ipcHandle);

    // Server loop: every 100ms read one element sequentially from the array
    int index = 0;
    float readValue = 0.0f;
    while (true) {
        checkCuda(hipMemcpy(&readValue, d_ptr + index, sizeof(float), hipMemcpyDeviceToHost),
                  "hipMemcpy read element");
        printf("Server: Read element at index %d = %.1f\n", index, readValue);
        
        index = (index + 1) % numElements; // Cycle through the array
        usleep(100000); // Sleep 100ms
    }

    // Clean-up (never reached)
    delete[] h_data;
    checkCuda(hipFree(d_ptr), "hipFree");
    return 0;
}
