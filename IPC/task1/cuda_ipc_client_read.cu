#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>

#define ITERATIONS 100

// Kernel that reads from src and writes to dst
__global__ void read_kernel(const float* src, float* dst, size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        dst[tid] = src[tid];
    }
}

bool file_exists(const char *filename) {
    struct stat buffer;
    return (stat(filename, &buffer) == 0);
}

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Client Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    FILE* fp_out = fopen("results.csv", "w");
    if (!fp_out) {
        perror("Client: fopen results.csv");
        exit(EXIT_FAILURE);
    }
    fprintf(fp_out, "Size_Bytes,IPC_Kernel_Time_ms,Local_Kernel_Time_ms\n");
    fflush(fp_out);

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "hipEventCreate start");
    checkCuda(hipEventCreate(&stop), "hipEventCreate stop");

    while (true) {
        if (file_exists("server_done.txt")) {
            printf("Client: Detected server completion. Exiting.\n");
            break;
        }

        if (!file_exists("ipc_handle.bin") || !file_exists("size.txt")) {
            usleep(100000); // 100ms
            continue;
        }

        size_t size_bytes;
        FILE* fp = fopen("size.txt", "r");
        if (!fp) {
            perror("Client: fopen size.txt");
            exit(EXIT_FAILURE);
        }
        fscanf(fp, "%zu", &size_bytes);
        fclose(fp);
        
        hipIpcMemHandle_t ipcHandle;
        fp = fopen("ipc_handle.bin", "rb");
        if (!fp) {
            perror("Client: fopen ipc_handle.bin");
            exit(EXIT_FAILURE);
        }
        fread(&ipcHandle, sizeof(ipcHandle), 1, fp);
        fclose(fp);

        float* ipc_d_ptr = nullptr;
        checkCuda(hipIpcOpenMemHandle((void**)&ipc_d_ptr, ipcHandle, hipIpcMemLazyEnablePeerAccess),
                  "hipIpcOpenMemHandle in client");

        size_t numElements = size_bytes / sizeof(float);
        
        // test1: IPC time
        float* d_dst_ipc = nullptr;
        checkCuda(hipMalloc(&d_dst_ipc, size_bytes), "hipMalloc d_dst_ipc in client");
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        read_kernel<<<blocksPerGrid, threadsPerBlock>>>(ipc_d_ptr, d_dst_ipc, numElements);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize after warm-up (IPC)"); // 一直不退出，直到读到第一个process
        float totalTime_ipc = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start, 0), "hipEventRecord start (IPC)");
            read_kernel<<<blocksPerGrid, threadsPerBlock>>>(ipc_d_ptr, d_dst_ipc, numElements);
            checkCuda(hipEventRecord(stop, 0), "hipEventRecord stop (IPC)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (IPC)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (IPC)");
            totalTime_ipc += ms;
        }
        float avgTime_ipc = totalTime_ipc / ITERATIONS;

        // printf("Client: Size: %zu bytes, IPC kernel avg time: %.6f ms, \n",
        //        size_bytes, avgTime_ipc);

        checkCuda(hipFree(d_dst_ipc), "hipFree d_dst_ipc in client");
        checkCuda(hipIpcCloseMemHandle(ipc_d_ptr), "hipIpcCloseMemHandle in client");

        // test2: local memory
        float* local_d_ptr = nullptr;
        checkCuda(hipMalloc(&local_d_ptr, size_bytes), "hipMalloc for local memory in client");
        checkCuda(hipMemset(local_d_ptr, 1, size_bytes), "hipMemset for local memory in client");
        float* d_dst_local = nullptr;
        checkCuda(hipMalloc(&d_dst_local, size_bytes), "hipMalloc d_dst_local in client");

        // warmup
        read_kernel<<<blocksPerGrid, threadsPerBlock>>>(local_d_ptr, d_dst_local, numElements);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize after warm-up (local)");

        float totalTime_local = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start, 0), "hipEventRecord start (local)");
            read_kernel<<<blocksPerGrid, threadsPerBlock>>>(local_d_ptr, d_dst_local, numElements);
            checkCuda(hipEventRecord(stop, 0), "hipEventRecord stop (local)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (local)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (local)");
            totalTime_local += ms;
        }
        float avgTime_local = totalTime_local / ITERATIONS;

        fprintf(fp_out, "%zu,%.6f,%.6f\n", size_bytes, avgTime_ipc, avgTime_local);
        fflush(fp_out);

        printf("Client: Size: %zu bytes, IPC kernel avg time: %.6f ms, Local kernel avg time: %.6f ms\n",
               size_bytes, avgTime_ipc, avgTime_local);


        
        checkCuda(hipFree(local_d_ptr), "hipFree local memory in client");
        checkCuda(hipFree(d_dst_local), "hipFree d_dst_local in client");
        

        fp = fopen("client_done.txt", "w");
        if (fp) {
            fprintf(fp, "done");
            fclose(fp);
        }

        remove("ipc_handle.bin");
        remove("size.txt");

        usleep(100000);
    }

    checkCuda(hipEventDestroy(start), "hipEventDestroy start in client");
    checkCuda(hipEventDestroy(stop), "hipEventDestroy stop in client");
    fclose(fp_out);

    return 0;
}
