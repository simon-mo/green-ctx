#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

#define SIZE (1024 * 1024)     // transmision size (4MB)
#define ITERATIONS 1000        // iteration times

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char* argv[]) {
    FILE* fp_out = fopen("results.csv", "w");
    if (!fp_out) {
        perror("fopen results.csv");
        exit(EXIT_FAILURE);
    }
    fprintf(fp_out, "Size_Bytes,IPC_Time_ms,Local_Time_ms\n");

    if (argc != 2) {
        printf("Usage: %s [server|client]\n", argv[0]);
        return 0;
    }

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "hipEventCreate start");
    checkCuda(hipEventCreate(&stop), "hipEventCreate stop");
    
    if (strcmp(argv[1], "server") == 0) {
        // Server
        float* d_ptr = nullptr;
        // init
        checkCuda(hipMalloc(&d_ptr, SIZE * sizeof(float)), "hipMalloc in server");
        checkCuda(hipMemset(d_ptr, 1, SIZE * sizeof(float)), "hipMemset in server");
        // IPC handle
        hipIpcMemHandle_t ipcHandle;
        checkCuda(hipIpcGetMemHandle(&ipcHandle, d_ptr), "hipIpcGetMemHandle");
        // write IPC to file
        FILE* fp = fopen("ipc_handle.bin", "wb");
        if (!fp) {
            perror("fopen");
            exit(EXIT_FAILURE);
        }
        fwrite(&ipcHandle, sizeof(ipcHandle), 1, fp);
        fclose(fp);
        
        printf("Server: IPC handle written to file. d_ptr = %p\n", d_ptr);
        printf("Server: Press Enter to exit...\n");
        getchar();
        
        checkCuda(hipFree(d_ptr), "hipFree in server");
        printf("Server: Memory freed and exiting.\n");
    } 
    else if (strcmp(argv[1], "client") == 0) {
        // Client
        hipIpcMemHandle_t ipcHandle;
        FILE* fp = fopen("ipc_handle.bin", "rb");
        if (!fp) {
            perror("fopen");
            exit(EXIT_FAILURE);
        }
        fread(&ipcHandle, sizeof(ipcHandle), 1, fp);
        fclose(fp);
        
        float* ipc_d_ptr = nullptr;
        checkCuda(hipIpcOpenMemHandle((void**)&ipc_d_ptr, ipcHandle, hipIpcMemLazyEnablePeerAccess),
                  "hipIpcOpenMemHandle");
        printf("Client: Opened IPC memory: %p\n", ipc_d_ptr);
        

        float* h_data = (float*)malloc(SIZE * sizeof(float));
        if (!h_data) {
            perror("malloc");
            exit(EXIT_FAILURE);
        }
        
        // CUDA event for timing
        hipEvent_t start, stop;
        checkCuda(hipEventCreate(&start), "hipEventCreate start");
        checkCuda(hipEventCreate(&stop), "hipEventCreate stop");
        // warm up
        checkCuda(hipMemcpy(h_data, ipc_d_ptr, SIZE * sizeof(float), hipMemcpyDeviceToHost),
                  "warm up memcpy (IPC)");
        
        // Result1: IPC memory copy
        float totalTime_ipc = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start), "hipEventRecord start (IPC)");
            checkCuda(hipMemcpy(h_data, ipc_d_ptr, SIZE * sizeof(float), hipMemcpyDeviceToHost),
                      "hipMemcpy (IPC)");
            checkCuda(hipEventRecord(stop), "hipEventRecord stop (IPC)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (IPC)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (IPC)");
            totalTime_ipc += ms;
        }
        float avgTime_ipc = totalTime_ipc / ITERATIONS;
        printf("Client: Average hipMemcpy time from IPC memory: %f ms\n", avgTime_ipc);
        
        // Result2: baseline
        float* local_d_ptr = nullptr;
        checkCuda(hipMalloc(&local_d_ptr, SIZE * sizeof(float)), "hipMalloc local");
        checkCuda(hipMemset(local_d_ptr, 1, SIZE * sizeof(float)), "hipMemset local");
        // warmup
        checkCuda(hipMemcpy(h_data, local_d_ptr, SIZE * sizeof(float), hipMemcpyDeviceToHost),
                  "warm up memcpy (local)");
        
        float totalTime_local = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start), "hipEventRecord start (local)");
            checkCuda(hipMemcpy(h_data, local_d_ptr, SIZE * sizeof(float), hipMemcpyDeviceToHost),
                      "hipMemcpy (local)");
            checkCuda(hipEventRecord(stop), "hipEventRecord stop (local)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (local)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (local)");
            totalTime_local += ms;
        }
        float avgTime_local = totalTime_local / ITERATIONS;
        printf("Client: Average hipMemcpy time from local memory: %f ms\n", avgTime_local);
        
        // clear
        checkCuda(hipEventDestroy(start), "hipEventDestroy start");
        checkCuda(hipEventDestroy(stop), "hipEventDestroy stop");
        free(h_data);
        checkCuda(hipIpcCloseMemHandle(ipc_d_ptr), "hipIpcCloseMemHandle");
        checkCuda(hipFree(local_d_ptr), "hipFree local");
        
        printf("Client: Tests completed. Exiting.\n");
    } 
    else {
        printf("Unknown mode: %s. Use 'server' or 'client'.\n", argv[1]);
    }
    
    return 0;
}
