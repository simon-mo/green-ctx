#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>

#define ITERATIONS 100

// Kernel that writes a constant value to every element of dst.
__global__ void write_kernel(float* dst, float value, size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        dst[tid] = value;
    }
}

// Check if a file exists
bool file_exists(const char *filename) {
    struct stat buffer;
    return (stat(filename, &buffer) == 0);
}

// Error checking function
void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Client Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Open results file for writing (CSV)
    FILE* fp_out = fopen("results_write.csv", "w");
    if (!fp_out) {
        perror("Client: fopen results_write.csv");
        exit(EXIT_FAILURE);
    }
    // CSV header: Memory size (bytes), IPC write kernel avg time (ms), Local write kernel avg time (ms)
    fprintf(fp_out, "Size_Bytes,IPC_Write_Kernel_Time_ms,Local_Write_Kernel_Time_ms\n");
    fflush(fp_out);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "hipEventCreate start");
    checkCuda(hipEventCreate(&stop), "hipEventCreate stop");

    while (true) {
        if (file_exists("server_done.txt")) {
            printf("Client: Detected server completion. Exiting.\n");
            break;
        }
        if (!file_exists("ipc_handle.bin") || !file_exists("size.txt")) {
            usleep(100000); // 100ms
            continue;
        }

        // Read the memory size from file "size.txt"
        size_t size_bytes;
        FILE* fp = fopen("size.txt", "r");
        if (!fp) {
            perror("Client: fopen size.txt");
            exit(EXIT_FAILURE);
        }
        fscanf(fp, "%zu", &size_bytes);
        fclose(fp);

        // Read the IPC handle from "ipc_handle.bin"
        hipIpcMemHandle_t ipcHandle;
        fp = fopen("ipc_handle.bin", "rb");
        if (!fp) {
            perror("Client: fopen ipc_handle.bin");
            exit(EXIT_FAILURE);
        }
        fread(&ipcHandle, sizeof(ipcHandle), 1, fp);
        fclose(fp);

        // Open the server-shared GPU memory using the IPC handle
        float* ipc_d_ptr = nullptr;
        checkCuda(hipIpcOpenMemHandle((void**)&ipc_d_ptr, ipcHandle, hipIpcMemLazyEnablePeerAccess),
                  "hipIpcOpenMemHandle in client");

        // Calculate the number of float elements
        size_t numElements = size_bytes / sizeof(float);

        // Determine grid and block dimensions for the kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // ---------------------- IPC Memory Write Kernel Test ----------------------
        // Pre-warm the kernel on IPC memory
        write_kernel<<<blocksPerGrid, threadsPerBlock>>>(ipc_d_ptr, 3.14f, numElements);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize after warm-up (IPC)");

        float totalTime_ipc = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start, 0), "hipEventRecord start (IPC)");
            write_kernel<<<blocksPerGrid, threadsPerBlock>>>(ipc_d_ptr, 3.14f, numElements);
            checkCuda(hipEventRecord(stop, 0), "hipEventRecord stop (IPC)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (IPC)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (IPC)");
            totalTime_ipc += ms;
        }
        float avgTime_ipc = totalTime_ipc / ITERATIONS;

        // ---------------------- Local Memory Write Kernel Test ----------------------
        // Allocate local GPU memory in the client
        float* local_d_ptr = nullptr;
        checkCuda(hipMalloc(&local_d_ptr, size_bytes), "hipMalloc for local memory in client");

        // Pre-warm the kernel on local memory
        write_kernel<<<blocksPerGrid, threadsPerBlock>>>(local_d_ptr, 3.14f, numElements);
        checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize after warm-up (local)");

        float totalTime_local = 0.0f;
        for (int i = 0; i < ITERATIONS; i++) {
            checkCuda(hipEventRecord(start, 0), "hipEventRecord start (local)");
            write_kernel<<<blocksPerGrid, threadsPerBlock>>>(local_d_ptr, 3.14f, numElements);
            checkCuda(hipEventRecord(stop, 0), "hipEventRecord stop (local)");
            checkCuda(hipEventSynchronize(stop), "hipEventSynchronize (local)");
            float ms = 0.0f;
            checkCuda(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime (local)");
            totalTime_local += ms;
        }
        float avgTime_local = totalTime_local / ITERATIONS;

        // Write the test results to CSV
        fprintf(fp_out, "%zu,%.6f,%.6f\n", size_bytes, avgTime_ipc, avgTime_local);
        fflush(fp_out);
        printf("Client: Size: %zu bytes, IPC write kernel avg time: %.6f ms, Local write kernel avg time: %.6f ms\n",
               size_bytes, avgTime_ipc, avgTime_local);

        // Free all allocated resources to minimize memory usage
        checkCuda(hipFree(local_d_ptr), "hipFree local memory in client");
        checkCuda(hipIpcCloseMemHandle(ipc_d_ptr), "hipIpcCloseMemHandle in client");

        // Signal completion for this test iteration
        fp = fopen("client_done.txt", "w");
        if (fp) {
            fprintf(fp, "done");
            fclose(fp);
        }
        remove("ipc_handle.bin");
        remove("size.txt");

        usleep(100000); // Delay before next iteration
    }

    checkCuda(hipEventDestroy(start), "hipEventDestroy start in client");
    checkCuda(hipEventDestroy(stop), "hipEventDestroy stop in client");
    fclose(fp_out);

    return 0;
}
