#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void polling_kernel(const double* shared, double* result) {
    const double eps = 1e-9;
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        while (fabs(shared[0] - 1.0) > eps) {
            __threadfence();
        }
        unsigned long long client_cycles = clock64();
        unsigned long long server_cycles = (unsigned long long) shared[1];
        unsigned long long latency_cycles = client_cycles - server_cycles;
        result[0] = shared[0];  
        result[1] = (double) latency_cycles;
        printf("Client kernel: Detected update. Server cycles = %llu, client cycles = %llu, latency cycles = %llu\n",
               server_cycles, client_cycles, latency_cycles);
    }
}

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Client Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

bool file_exists(const char* filename) {
    struct stat buffer;
    return (stat(filename, &buffer) == 0);
}

int main() {
    while (!file_exists("ipc_handle.bin")) {
        usleep(100000);  
    }

    hipIpcMemHandle_t ipcHandle;
    FILE* fp = fopen("ipc_handle.bin", "rb");
    if (!fp) {
        perror("Client: fopen ipc_handle.bin");
        exit(EXIT_FAILURE);
    }
    fread(&ipcHandle, sizeof(ipcHandle), 1, fp);
    fclose(fp);
    printf("Client: IPC handle read from file\n");


    double* d_shared = nullptr;
    checkCuda(hipIpcOpenMemHandle((void**)&d_shared, ipcHandle, hipIpcMemLazyEnablePeerAccess),
              "hipIpcOpenMemHandle");
    printf("Client: Shared GPU memory opened at %p\n", d_shared);

    double* d_result = nullptr;
    checkCuda(hipMalloc(&d_result, 2 * sizeof(double)), "hipMalloc d_result");

    polling_kernel<<<1, 1>>>(d_shared, d_result);
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    double* h_result = (double*)malloc(2 * sizeof(double));
    if (!h_result) {
        fprintf(stderr, "Client Error: malloc failed\n");
        exit(EXIT_FAILURE);
    }
    checkCuda(hipMemcpy(h_result, d_result, 2 * sizeof(double), hipMemcpyDeviceToHost),
              "hipMemcpy d_result to h_result");
    unsigned int clock_rate_khz = 0;
    checkCuda(hipDeviceGetAttribute((int*)&clock_rate_khz, hipDeviceAttributeClockRate, 0),
              "hipDeviceGetAttribute");
    double us_per_cycle = 1000.0 / clock_rate_khz;
    double latency_us = h_result[1] * us_per_cycle;
    printf("Client: Detected update. Value = %.0f, latency cycles = %.0f, latency = %.2f us\n",
           h_result[0], h_result[1], latency_us);
    checkCuda(hipIpcCloseMemHandle(d_shared), "hipIpcCloseMemHandle");
    checkCuda(hipFree(d_result), "hipFree d_result");
    free(h_result);

    return 0;
}