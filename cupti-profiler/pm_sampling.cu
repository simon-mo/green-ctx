#include "hip/hip_runtime.h"
/*
 * Copyright 2024 NVIDIA Corporation. All rights reserved
 *
 * This sample demonstrates the usage of the PM sampling feature in the CUDA
 * Profiling Tools Interface (CUPTI). The implementation consists of three main
 * components:
 *
 * 1. Performance Metric Sampling
 *    - Query available metrics and their properties
 *    - Configure and collect PM sampling data for CUDA workloads
 *
 * 2. Workload Management
 *    - Main thread: Launches CUDA workload (vector addition kernel)
 *    - Decode thread: Processes collected sampling data
 *
 * 3. Data Collection
 *    - Continuous collection using separate threads
 *    - Hardware buffer management and data decoding
 *    - Sample evaluation and metric calculation
 */

#include <atomic>
#include <chrono>
#include <sstream>
#include <stdio.h>
#include <string.h>
#include <thread>

#ifdef _WIN32
#define strdup _strdup
#endif

// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "pm_sampling.h"

// Global control flag for decode thread
std::atomic<bool> stopDecodeThread(false);

/**
 * @brief Simple vector addition kernel for workload generation
 * @param pA First input vector
 * @param pB Second input vector
 * @param pC Output vector
 * @param N Vector size
 */
__global__ void vectorAdd(const int *pA, const int *pB, int *pC, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    pC[i] = pA[i] + pB[i];
  }
}

/**
 * @brief Class managing the vector addition workload for sampling
 *
 * Handles memory allocation, data initialization, kernel execution,
 * and result verification for the vector addition operation.
 */
class VectorLaunchWorkLoad {
public:
  /**
   * @brief Constructor initializing workload parameters
   * @param numElements Number of vector elements
   * @param threadsPerBlock Number of threads per block
   */
  VectorLaunchWorkLoad(int numElements = 50000, int threadsPerBlock = 256)
      : m_numOfElements(numElements), m_threadsPerBlock(threadsPerBlock) {
    m_size = m_numOfElements * sizeof(int);
    m_blocksPerGrid =
        (m_numOfElements + m_threadsPerBlock - 1) / m_threadsPerBlock;
    pHostA.resize(m_numOfElements);
    pHostB.resize(m_numOfElements);
    pHostC.resize(m_numOfElements);
  }

  ~VectorLaunchWorkLoad() {}

  /**
   * @brief Initialize vector with sequential values
   */
  void InitializeVector(std::vector<int> &pVector) {
    for (int i = 0; i < m_numOfElements; i++) {
      pVector[i] = i;
    }
  }

  /**
   * @brief Free allocated device memory
   */
  void CleanUp() {
    RUNTIME_API_CALL(hipFree(pDeviceA));
    RUNTIME_API_CALL(hipFree(pDeviceB));
    RUNTIME_API_CALL(hipFree(pDeviceC));
  }

  /**
   * @brief Set up vectors and allocate device memory
   */
  void SetUp() {
    // Initialize input vectors
    InitializeVector(pHostA);
    InitializeVector(pHostB);
    std::fill(pHostC.begin(), pHostC.end(), 0);

    // Allocate device memory
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, m_size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, m_size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, m_size));

    // Copy to device
    RUNTIME_API_CALL(
        hipMemcpy(pDeviceA, pHostA.data(), m_size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(
        hipMemcpy(pDeviceB, pHostB.data(), m_size, hipMemcpyHostToDevice));
  }

  /**
   * @brief Verify results and clean up resources
   */
  void TearDown() {
    RUNTIME_API_CALL(
        hipMemcpy(pHostC.data(), pDeviceC, m_size, hipMemcpyDeviceToHost));

    // Verify results
    for (int i = 0; i < m_numOfElements; ++i) {
      int sum = pHostA[i] + pHostB[i];
      if (pHostC[i] != sum) {
        fprintf(stderr, "Error: Result verification failed.\n");
        exit(EXIT_FAILURE);
      }
    }
    printf("Result verification passed.\n");
    CleanUp();
  }

  /**
   * @brief Launch vector addition kernel
   * @return CUDA error code
   */
  hipError_t LaunchKernel() {
    vectorAdd<<<m_blocksPerGrid, m_threadsPerBlock>>>(
        pDeviceA, pDeviceB, pDeviceC, m_numOfElements);
    return hipGetLastError();
  }

private:
  int m_numOfElements;
  int m_threadsPerBlock;
  int m_blocksPerGrid;
  size_t m_size;

  int *pDeviceA, *pDeviceB, *pDeviceC;
  std::vector<int> pHostA, pHostB, pHostC;
};

/**
 * @brief Command line argument structure
 */
struct ParsedArgs {
  int deviceIndex = 0;
  int queryBaseMetrics = 0;
  int queryMetricProperties = 0;
  std::string chipName;
  uint64_t samplingInterval = 100000;            // 100us
  size_t hardwareBufferSize = 512 * 1024 * 1024; // 512MB
  uint64_t maxSamples = 10000;
  std::vector<const char *> metrics = {
      "gr__cycles_active.avg",                           // Active Cycles
      "gr__cycles_elapsed.max",                          // Elapsed Cycles
      "gpu__time_duration.sum",                          // Duration
      "sm__inst_executed_realtime.avg.per_cycle_active", // Inst Executed per
                                                         // Active Cycle
      "sm__cycles_active.avg"                            // SM Active Cycles
  };
};

// Forward declarations
ParsedArgs parseArgs(int argc, char *argv[]);
void PmSamplingDeviceSupportStatus(hipDevice_t device);
int PmSamplingCollection(std::vector<uint8_t> &counterAvailibilityImage,
                         ParsedArgs &args);
int PmSamplingQueryMetrics(std::string chipName,
                           std::vector<uint8_t> &counterAvailibilityImage,
                           ParsedArgs &args);
void DecodeCounterData(std::vector<uint8_t> &counterDataImage,
                       std::vector<const char *> metricsList,
                       CuptiPmSampling &cuptiPmSamplingTarget,
                       CuptiProfilerHost &pmSamplingHost, CUptiResult &result);

/**
 * @brief Main entry point for the PM sampling example
 */
int main(int argc, char *argv[]) {
  ParsedArgs args = parseArgs(argc, argv);
  DRIVER_API_CALL(hipInit(0));

  std::string chipName = args.chipName;
  std::vector<uint8_t> counterAvailibilityImage;

  // Get device information and check support
  if (args.deviceIndex >= 0) {
    hipDevice_t cuDevice;
    DRIVER_API_CALL(hipDeviceGet(&cuDevice, args.deviceIndex));
    PmSamplingDeviceSupportStatus(cuDevice);

    CuptiPmSampling::GetChipName(args.deviceIndex, chipName);
    CuptiPmSampling::GetCounterAvailabilityImage(args.deviceIndex,
                                                 counterAvailibilityImage);
  }

  // Handle query requests or run sampling collection
  if (args.queryBaseMetrics || args.queryMetricProperties) {
    return PmSamplingQueryMetrics(chipName, counterAvailibilityImage, args);
  } else {
    return PmSamplingCollection(counterAvailibilityImage, args);
  }
}

/**
 * @brief Query and display metric information
 *
 * @param chipName Name of the GPU chip
 * @param counterAvailibilityImage Counter availability data
 * @param args Command line arguments
 * @return Status code
 */
int PmSamplingQueryMetrics(std::string chipName,
                           std::vector<uint8_t> &counterAvailibilityImage,
                           ParsedArgs &args) {
  CuptiProfilerHost pmSamplingHost;
  pmSamplingHost.SetUp(chipName, counterAvailibilityImage);

  if (args.queryBaseMetrics) {
    std::vector<std::string> baseMetrics;
    CUPTI_API_CALL(pmSamplingHost.GetSupportedBaseMetrics(baseMetrics));
    printf("Base Metrics:\n");
    for (const auto &metric : baseMetrics) {
      printf("  %s\n", metric.c_str());
    }
    return 0;
  }

  if (args.queryMetricProperties) {
    for (const auto &metricName : args.metrics) {
      std::vector<std::string> subMetrics;
      CUPTI_API_CALL(pmSamplingHost.GetSubMetrics(metricName, subMetrics));
      printf("Sub Metrics for %s:\n", metricName);
      for (const auto &metric : subMetrics) {
        printf("  %s\n", metric.c_str());
      }

      std::string metricDescription;
      CUpti_MetricType metricType;
      CUPTI_API_CALL(pmSamplingHost.GetMetricProperties(metricName, metricType,
                                                        metricDescription));

      printf("Metric Description: %s\n", metricDescription.c_str());
      printf("Metric Type: %s\n",
             metricType == CUPTI_METRIC_TYPE_COUNTER   ? "Counter"
             : (metricType == CUPTI_METRIC_TYPE_RATIO) ? "Ratio"
                                                       : "Throughput");
      printf("\n");
    }
    return 0;
  }

  pmSamplingHost.TearDown();
  return 0;
}

/**
 * @brief Run PM sampling collection on a workload
 *
 * @param counterAvailibilityImage Counter availability data
 * @param args Command line arguments
 * @return Status code
 */
int PmSamplingCollection(std::vector<uint8_t> &counterAvailibilityImage,
                         ParsedArgs &args) {
  // Initialize profiling components
  std::string chipName;
  CuptiPmSampling::GetChipName(args.deviceIndex, chipName);

  CuptiProfilerHost pmSamplingHost;
  pmSamplingHost.SetUp(chipName, counterAvailibilityImage);

  std::vector<uint8_t> configImage;
  CUPTI_API_CALL(pmSamplingHost.CreateConfigImage(args.metrics, configImage));

  CuptiPmSampling cuptiPmSamplingTarget;
  cuptiPmSamplingTarget.SetUp(args.deviceIndex);

  // Configure and start sampling
  CUPTI_API_CALL(cuptiPmSamplingTarget.EnablePmSampling(args.deviceIndex));
  CUPTI_API_CALL(cuptiPmSamplingTarget.SetConfig(
      configImage, args.hardwareBufferSize, args.samplingInterval));

  // Create counter data storage
  std::vector<uint8_t> counterDataImage;
  CUPTI_API_CALL(cuptiPmSamplingTarget.CreateCounterDataImage(
      args.maxSamples, args.metrics, counterDataImage));

  // Set up workload
  VectorLaunchWorkLoad vectorWorkLoad;
  vectorWorkLoad.SetUp();

  // Start decode thread
  CUptiResult threadFuncResult;
  std::thread decodeThread(
      DecodeCounterData, std::ref(counterDataImage), std::ref(args.metrics),
      std::ref(cuptiPmSamplingTarget), std::ref(pmSamplingHost),
      std::ref(threadFuncResult));

  auto joinDecodeThread = [&]() {
    stopDecodeThread = true;
    decodeThread.join();
    if (threadFuncResult != CUPTI_SUCCESS) {
      const char *errstr;
      cuptiGetResultString(threadFuncResult, &errstr);
      std::cerr << "DecodeCounterData Thread failed with error " << errstr
                << std::endl;
      return 1;
    }
    return 0;
  };

  // Run workload with sampling
  CUPTI_API_CALL(cuptiPmSamplingTarget.StartPmSampling());
  stopDecodeThread = false;

  const size_t NUM_OF_ITERATIONS = 100000;
  for (size_t ii = 0; ii < NUM_OF_ITERATIONS; ++ii) {
    hipError_t result = vectorWorkLoad.LaunchKernel();
    if (result != hipSuccess) {
      std::cerr << "Kernel launch failed " << hipGetErrorString(result)
                << std::endl;
      return joinDecodeThread();
    }
  }

  hipError_t errResult = hipDeviceSynchronize();
  if (errResult != hipSuccess) {
    std::cerr << "DeviceSync Failed " << hipGetErrorString(errResult)
              << std::endl;
    return joinDecodeThread();
  }

  // Clean up
  CUPTI_API_CALL(cuptiPmSamplingTarget.StopPmSampling());
  joinDecodeThread();

  pmSamplingHost.PrintSampleRanges();
  CUPTI_API_CALL(cuptiPmSamplingTarget.DisablePmSampling());

  cuptiPmSamplingTarget.TearDown();
  pmSamplingHost.TearDown();
  vectorWorkLoad.TearDown();
  return 0;
}

/**
 * @brief Thread function for decoding PM sampling data
 *
 * Continuously decodes sampling data until stopped, evaluating metrics
 * for each completed sample.
 *
 * @param counterDataImage Buffer for counter data
 * @param metricsList List of metrics to evaluate
 * @param cuptiPmSamplingTarget PM sampling target
 * @param pmSamplingHost Profiler host
 * @param result Output parameter for operation result
 */
void DecodeCounterData(std::vector<uint8_t> &counterDataImage,
                       std::vector<const char *> metricsList,
                       CuptiPmSampling &cuptiPmSamplingTarget,
                       CuptiProfilerHost &pmSamplingHost, CUptiResult &result) {
  while (!stopDecodeThread) {
    // Decode collected data
    const char *errstr;
    result = cuptiPmSamplingTarget.DecodePmSamplingData(counterDataImage);
    if (result != CUPTI_SUCCESS) {
      cuptiGetResultString(result, &errstr);
      std::cerr << "DecodePmSamplingData failed with error " << errstr
                << std::endl;
      return;
    }

    // Get completed sample info
    CUpti_PmSampling_GetCounterDataInfo_Params counterDataInfo{
        CUpti_PmSampling_GetCounterDataInfo_Params_STRUCT_SIZE};
    counterDataInfo.pCounterDataImage = counterDataImage.data();
    counterDataInfo.counterDataImageSize = counterDataImage.size();
    result = cuptiPmSamplingGetCounterDataInfo(&counterDataInfo);
    if (result != CUPTI_SUCCESS) {
      cuptiGetResultString(result, &errstr);
      std::cerr << "cuptiPmSamplingGetCounterDataInfo failed with error "
                << errstr << std::endl;
      return;
    }

    // Process completed samples
    for (size_t sampleIndex = 0;
         sampleIndex < counterDataInfo.numCompletedSamples; ++sampleIndex) {
      pmSamplingHost.EvaluateCounterData(
          cuptiPmSamplingTarget.GetPmSamplerObject(), sampleIndex, metricsList,
          counterDataImage);
    }

    // Reset for next batch
    result = cuptiPmSamplingTarget.ResetCounterDataImage(counterDataImage);
    if (result != CUPTI_SUCCESS) {
      cuptiGetResultString(result, &errstr);
      std::cerr << "ResetCounterDataImage failed with error " << errstr
                << std::endl;
      return;
    }
  }
}

/**
 * @brief Print command-line usage information
 */
void PrintHelp() {
  printf("Usage:\n");
  printf("  Query Metrics:\n");
  printf("    List Base Metrics : ./pm_sampling --device/-d <deviceIndex> "
         "--chip/-c <chipname> --queryBaseMetrics/-q\n");
  printf("    List submetrics   : ./pm_sampling --device/-d <deviceIndex> "
         "--chip/-c <chipname> --metrics/-m <metric1,metric2,...> "
         "--queryMetricsProp/-p\n");
  printf("  Note: when device index flag is passed, the chip name flag will be "
         "ignored.\n");
  printf("  PM Sampling:\n");
  printf("    Collection: ./pm_sampling --device/-d <deviceIndex> "
         "--samplingInterval/-i <samplingInterval> "
         "--maxsamples/-s <maxSamples in CounterDataImage> "
         "--hardwareBufferSize/-b <hardware buffer size> "
         "--metrics/-m <metric1,metric2,...>\n");
}

/**
 * @brief Parse command line arguments
 *
 * @param argc Argument count
 * @param argv Argument values
 * @return ParsedArgs structure containing parsed arguments
 */
ParsedArgs parseArgs(int argc, char *argv[]) {
  ParsedArgs args;
  for (int i = 1; i < argc; i++) {
    std::string arg = argv[i];
    if (arg == "--device" || arg == "-d") {
      args.deviceIndex = std::stoi(argv[++i]);
    } else if (arg == "--samplingInterval" || arg == "-i") {
      args.samplingInterval = std::stoull(argv[++i]);
    } else if (arg == "--maxsamples" || arg == "-s") {
      args.maxSamples = std::stoull(argv[++i]);
    } else if (arg == "--hardwareBufferSize" || arg == "-b") {
      args.hardwareBufferSize = std::stoull(argv[++i]);
    } else if (arg == "--chip" || arg == "-c") {
      args.chipName = std::string(argv[++i]);
    } else if (arg == "--queryBaseMetrics" || arg == "-q") {
      args.queryBaseMetrics = 1;
    } else if (arg == "--queryMetricsProp" || arg == "-p") {
      args.queryMetricProperties = 1;
    } else if (arg == "--metrics" || arg == "-m") {
      std::stringstream ss(argv[++i]);
      std::string metric;
      args.metrics.clear();
      while (std::getline(ss, metric, ',')) {
        args.metrics.push_back(strdup(metric.c_str()));
      }
    } else if (arg == "--help" || arg == "-h") {
      PrintHelp();
      exit(EXIT_SUCCESS);
    } else {
      fprintf(stderr, "Invalid argument: %s\n", arg.c_str());
      PrintHelp();
      exit(EXIT_FAILURE);
    }
  }
  return args;
}

/**
 * @brief Check if PM sampling is supported on the device
 *
 * Verifies various device capabilities and configurations that may
 * affect PM sampling support.
 *
 * @param device CUDA device to check
 */
void PmSamplingDeviceSupportStatus(hipDevice_t device) {
  CUpti_Profiler_DeviceSupported_Params params = {
      CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE};
  params.cuDevice = device;
  params.api = CUPTI_PROFILER_PM_SAMPLING;
  CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

  if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED) {
    ::std::cerr << "Unable to profile on device " << device << ::std::endl;

    if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr << "\tdevice architecture is not supported" << ::std::endl;
    }
    if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr << "\tdevice sli configuration is not supported"
                  << ::std::endl;
    }
    if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr << "\tdevice vgpu configuration is not supported"
                  << ::std::endl;
    } else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED) {
      ::std::cerr << "\tdevice vgpu configuration disabled profiling support"
                  << ::std::endl;
    }
    if (params.confidentialCompute ==
        CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr
          << "\tdevice confidential compute configuration is not supported"
          << ::std::endl;
    }
    if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported"
                  << ::std::endl;
    }
    if (params.wsl == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) {
      ::std::cerr << "\tWSL is not supported" << ::std::endl;
    }

    exit(EXIT_WAIVED);
  }
}
